#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert> // ???
#include <iostream>
#include <cmath>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int N)
{
  // blockDim: size of the thread block
	// blockIdx : id of the block
	// threadIdx: offset of the thread inside the block
	// Calculate global thread thread ID
	int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

	// Boundary check
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid]
	}
}

int main()
{
	// Array size of 2^16 
	const int N = pow(2, 16);
	size_t bytes = N * sizeof(int);

	// Declare unified memory pointers
	int *a, *b, *c;

	// Allocation memory for these pointers
	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);
	
	// Initialize vectors
	for(int i=0; i<N; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    // Threads per CTA
    int BLOCK_SIZE = pow(2, 10);

    // CTAs per Grid
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Call CUDA kernel
    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a,b,c,N);

    // Wait for all previous operations before using values
    hipDeviceSynchronize();

    // Verify the result on the CPU
    for(int i=0; i<N; i++)
    {
        assert(c[i] == a[i] + b[i]);
    }

    // Free unified memory(same as memory allocated with hipMalloc)
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}


